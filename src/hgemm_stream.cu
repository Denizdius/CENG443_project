#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_cooperative_groups.h>
#include <mma.h>
#include <stdio.h>

// Enable Tensor Core operations
#pragma enable_tf32_tensor_core_optimization

#define N 1024
#define BLOCK_SIZE 16
#define WARP_SIZE 32

// WMMA matrix tiles
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Error checking macro
#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(2); \
    } \
}

// Add non-Tensor Core version of HGEMM
__global__ void hgemm_normal(const half* A, const half* B, float* C, int start_idx, int chunk_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Adjust indices based on chunk
    row += start_idx;
    
    if (row < start_idx + chunk_size && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += __half2float(A[row * N + k]) * __half2float(B[k * N + col]);
        }
        C[row * N + col] = sum;
    }
}

__global__ void hgemm_tensor_core(const half* A, const half* B, float* C, int start_idx, int chunk_size) {
    // Each warp computes a 16x16 output tile
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int warpN = blockIdx.y;

    // Adjust warpM based on chunk
    warpM += start_idx / WMMA_M;

    // Check if this warp should process this chunk
    if (warpM * WMMA_M >= start_idx + chunk_size) {
        return;
    }

    // Declare the fragments
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;

    // Initialize the output to zero
    nvcuda::wmma::fill_fragment(acc_frag, 0.0f);

    // Load and multiply
    for (int k = 0; k < N; k += WMMA_K) {
        const half* a_tile = A + (warpM * WMMA_M) * N + k;
        const half* b_tile = B + k * N + warpN * WMMA_N;
        
        nvcuda::wmma::load_matrix_sync(a_frag, a_tile, N);
        nvcuda::wmma::load_matrix_sync(b_frag, b_tile, N);
        nvcuda::wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    // Store the output
    float* c_tile = C + (warpM * WMMA_M) * N + warpN * WMMA_N;
    nvcuda::wmma::store_matrix_sync(c_tile, acc_frag, N, nvcuda::wmma::mem_row_major);
}

int main() {
    // Ensure matrix dimensions are compatible with WMMA
    static_assert(N % WMMA_M == 0, "Matrix size must be divisible by WMMA_M");
    static_assert(N % WMMA_N == 0, "Matrix size must be divisible by WMMA_N");
    static_assert(N % WMMA_K == 0, "Matrix size must be divisible by WMMA_K");

    half *a_h, *b_h;
    float *c_h;
    half *a_d, *b_d;
    float *c_d;

    // Allocate host memory
    a_h = new half[N * N];
    b_h = new half[N * N];
    c_h = new float[N * N];

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        a_h[i] = __float2half(1.0f);
        b_h[i] = __float2half(1.0f);
    }

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&a_d, N * N * sizeof(half)));
    CUDA_CHECK(hipMalloc(&b_d, N * N * sizeof(half)));
    CUDA_CHECK(hipMalloc(&c_d, N * N * sizeof(float)));
    
    // Additional output buffers for concurrent execution
    float *c_stream1_d, *c_stream2_d;
    CUDA_CHECK(hipMalloc(&c_stream1_d, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&c_stream2_d, N * N * sizeof(float)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(a_d, a_h, N * N * sizeof(half), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(b_d, b_h, N * N * sizeof(half), hipMemcpyHostToDevice));

    // Create CUDA streams
    hipStream_t stream_tensor, stream_normal;
    CUDA_CHECK(hipStreamCreate(&stream_tensor));
    CUDA_CHECK(hipStreamCreate(&stream_normal));

    // Setup timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Divide the matrix into two parts (ensure it's WMMA aligned)
    const int chunk_size = (N / 2 / WMMA_M) * WMMA_M;

    // Launch configurations
    dim3 gridDim_tensor((chunk_size + WMMA_M - 1) / WMMA_M, N / WMMA_N);
    dim3 blockDim_tensor(WARP_SIZE, 1);

    dim3 gridDim_normal((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (chunk_size + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockDim_normal(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim_tensor_full((N + WMMA_M - 1) / WMMA_M, N / WMMA_N);
    dim3 gridDim_normal_full((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    float ms_normal = 0, ms_tensor = 0, ms_stream = 0;

    // Clear all output buffers
    CUDA_CHECK(hipMemset(c_d, 0, N * N * sizeof(float)));
    CUDA_CHECK(hipMemset(c_stream1_d, 0, N * N * sizeof(float)));
    CUDA_CHECK(hipMemset(c_stream2_d, 0, N * N * sizeof(float)));

    // Test Case 1: Two Tensor Core kernels in different streams
    printf("\n=== Test Case 1: Two Tensor Core Kernels ===\n");
    CUDA_CHECK(hipEventRecord(start));
    
    hgemm_tensor_core<<<gridDim_tensor, blockDim_tensor, 0, stream_tensor>>>(a_d, b_d, c_stream1_d, 0, chunk_size);

    hgemm_tensor_core<<<gridDim_tensor, blockDim_tensor, 0, stream_normal>>>(a_d, b_d, c_stream2_d, chunk_size, chunk_size);
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms_stream, start, stop));
    printf("Two Tensor Core kernels time: %.3f ms\n", ms_stream);

    // Combine results from both streams
    hipMemcpyAsync(c_d, c_stream1_d, chunk_size * N * sizeof(float), hipMemcpyDeviceToDevice, stream_tensor);
    hipMemcpyAsync(c_d + chunk_size * N, c_stream2_d + chunk_size * N, chunk_size * N * sizeof(float), hipMemcpyDeviceToDevice, stream_normal);
    hipDeviceSynchronize();

    // Clear buffers for next test
    CUDA_CHECK(hipMemset(c_stream1_d, 0, N * N * sizeof(float)));
    CUDA_CHECK(hipMemset(c_stream2_d, 0, N * N * sizeof(float)));

    // Test Case 2: Two Normal CUDA kernels in different streams
    printf("\n=== Test Case 2: Two Normal CUDA Kernels ===\n");
    CUDA_CHECK(hipEventRecord(start));
    
    hgemm_normal<<<gridDim_normal, blockDim_normal, 0, stream_tensor>>>(
        a_d, b_d, c_stream1_d, 0, chunk_size);
        
    hgemm_normal<<<gridDim_normal, blockDim_normal, 0, stream_normal>>>(
        a_d, b_d, c_stream2_d, chunk_size, chunk_size);
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms_stream, start, stop));
    printf("Two Normal CUDA kernels time: %.3f ms\n", ms_stream);

    // Combine results
    hipMemcpyAsync(c_d, c_stream1_d, chunk_size * N * sizeof(float), hipMemcpyDeviceToDevice, stream_tensor);
    hipMemcpyAsync(c_d + chunk_size * N, c_stream2_d + chunk_size * N, chunk_size * N * sizeof(float), hipMemcpyDeviceToDevice, stream_normal);
    hipDeviceSynchronize();

    // Clear buffers for next test
    CUDA_CHECK(hipMemset(c_stream1_d, 0, N * N * sizeof(float)));
    CUDA_CHECK(hipMemset(c_stream2_d, 0, N * N * sizeof(float)));

    // Test Case 3: Mixed Tensor Core and Normal CUDA kernels
    printf("\n=== Test Case 3: Mixed Tensor Core and Normal CUDA Kernels ===\n");
    CUDA_CHECK(hipEventRecord(start));
    
    hgemm_tensor_core<<<gridDim_tensor, blockDim_tensor, 0, stream_tensor>>>(
        a_d, b_d, c_stream1_d, 0, chunk_size);
            
    hgemm_normal<<<gridDim_normal, blockDim_normal, 0, stream_normal>>>(
        a_d, b_d, c_stream2_d, chunk_size, chunk_size);
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms_stream, start, stop));
    printf("Mixed kernels time: %.3f ms\n", ms_stream);

    // Combine results
    hipMemcpyAsync(c_d, c_stream1_d, chunk_size * N * sizeof(float), hipMemcpyDeviceToDevice, stream_tensor);
    hipMemcpyAsync(c_d + chunk_size * N, c_stream2_d + chunk_size * N, chunk_size * N * sizeof(float), hipMemcpyDeviceToDevice, stream_normal);
    hipDeviceSynchronize();

    // Print comparative results
    printf("\n=== Performance Comparison ===\n");
    printf("Matrix size: %dx%d\n", N, N);
    printf("1. Normal CUDA cores (full matrix): %.3f ms\n", ms_normal);
    printf("2. Tensor cores (full matrix): %.3f ms\n", ms_tensor);
    printf("3. Streamed version (half tensor + half normal): %.3f ms\n", ms_stream);
    printf("\nSpeedup Analysis:\n");
    printf("Tensor vs Normal: %.2fx\n", ms_normal / ms_tensor);
    printf("Streamed vs Normal: %.2fx\n", ms_normal / ms_stream);
    printf("Streamed vs Tensor: %.2fx\n", ms_tensor / ms_stream);

    // Verify results
    float *c_verify_h = new float[N * N];
    CUDA_CHECK(hipMemcpy(c_verify_h, c_d, N * N * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("\n=== Result Verification ===\n");
    printf("C[0][0] = %.0f\n", c_verify_h[0]);
    printf("C[%d][0] = %.0f\n", chunk_size, c_verify_h[chunk_size * N]);
    printf("Expected value = %d\n", N);

    // Cleanup
    delete[] c_verify_h;
    delete[] a_h;
    delete[] b_h;
    delete[] c_h;
    
    CUDA_CHECK(hipFree(a_d));
    CUDA_CHECK(hipFree(b_d));
    CUDA_CHECK(hipFree(c_d));
    CUDA_CHECK(hipFree(c_stream1_d));
    CUDA_CHECK(hipFree(c_stream2_d));
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipStreamDestroy(stream_tensor));
    CUDA_CHECK(hipStreamDestroy(stream_normal));

    return 0;
}
